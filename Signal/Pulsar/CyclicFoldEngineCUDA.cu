#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2012 by Glenn Jones and Paul Demorest
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

//#define _DEBUG 1

//#define SET_BINS_GPU 1

#include "dsp/CyclicFoldEngineCUDA.h"
#include "dsp/MemoryCUDA.h"

#include "Error.h"
#include "debug.h"

#include <memory>
#include <fstream>

using namespace std;

CUDA::CyclicFoldEngineCUDA::CyclicFoldEngineCUDA (hipStream_t _stream)
{
  lagbinplan = NULL;
  d_binplan = NULL;
  d_lagdata = NULL;
  current_turn = 0;
  use_set_bins = true;


  // no data on either the host or device
  synchronized = true;

  stream = _stream;
}

CUDA::CyclicFoldEngineCUDA::~CyclicFoldEngineCUDA ()
{
  
  if (lagbinplan) {
    cerr << "CUDA::CyclicFoldEngineCUDA::~CyclicFoldEngineCUDA freeing lagbinplan" <<endl;
    delete [] lagbinplan;
  }
  if (d_binplan) {
    cerr << "CUDA::CyclicFoldEngineCUDA::~CyclicFoldEngineCUDA freeing d_binplan" <<endl;
    hipFree(d_binplan);
  }
  if (d_lagdata) {
    cerr << "CUDA::CyclicFoldEngineCUDA::~CyclicFoldEngineCUDA freeing d_lagdata" <<endl;
    hipFree(d_lagdata);
  }
  cerr << "CUDA::CyclicFoldEngineCUDA::~CyclicFoldEngineCUDA finished" <<endl;
}

void CUDA::CyclicFoldEngineCUDA::synch (dsp::PhaseSeries *out) try
{

  if (dsp::Operation::verbose)
    cerr << "CUDA::CyclicFoldEngineCUDA::synch this=" << this << endl;

  if (synchronized)
    return;

  if (dsp::Operation::verbose)
    cerr << "CUDA::CyclicFoldEngineCUDA::synch output=" << output << endl;

  cerr << "CUDA::CyclicFoldEngineCUDA::synch out=" << out <<" out.ndat_folded=" << out->get_ndat_folded()
		 <<endl; // << " output.ndatfolded" << output->get_ndat_folded() << endl;

  // transfer lag data from GPU

  hipError_t error;
  if (stream)
    error = hipMemcpyAsync (lagdata,d_lagdata,lagdata_size*sizeof(float),hipMemcpyDeviceToHost,stream);
  else
    error = hipMemcpy (lagdata,d_lagdata,lagdata_size*sizeof(float),hipMemcpyDeviceToHost);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::sync",
                 "hipMemcpy%s %s", 
                 stream?"Async":"", hipGetErrorString (error));

  // Call usual synch() to do transform
  dsp::CyclicFoldEngine::synch(out);

  synchronized = true;
}
catch (Error& error)
{
  throw error += "CUDA::CyclicFoldEngineCUDA::synch";
}

void CUDA::CyclicFoldEngineCUDA::set_ndat (uint64_t _ndat, uint64_t _idat_start)
{
	// ndat is idat_end - idat_start
	// binplan_size is _ndat
  setup();

  if (parent->verbose)
    cerr << "dsp::CyclicFoldEngine::set_ndat ndat=" << _ndat << endl;
/*// FOllowing moved to set_bins
  if (_ndat > binplan_size) {

    if (parent->verbose)
      cerr << "dsp::CyclicFoldEngine::set_ndat alloc binplan" << endl;

    if (lagbinplan) {
      delete [] lagbinplan;
    }
    lagbinplan = new bin [_ndat*nlag];


    binplan_size = _ndat;
  }
  memset(lagbinplan, 0 , sizeof(bin)*_ndat*nlag);
  */
  current_turn = 0;
  last_ibin = 0;
  ndat_fold = _ndat;
  idat_start = _idat_start;

  if (parent->verbose)
    cerr << "dsp::CyclicFoldEngine::set_ndat "
      << "nlag=" << nlag << " "
      << "nbin=" << nbin << " "
      << "npol=" << npol_out << " "
      << "nchan=" << nchan << endl;

  uint64_t _lagdata_size = nlag * nbin * npol_out * ndim * nchan;

  if (parent->verbose)
    cerr << "dsp::CyclicFoldEngine::set_ndat lagdata_size=" << _lagdata_size << endl;

  if (_lagdata_size > lagdata_size) {
    if (parent->verbose)
      cerr << "dsp::CyclicFoldEngine::set_ndat alloc lagdata" << endl;
    if (lagdata) delete [] lagdata;
    lagdata = new float [_lagdata_size];
    lagdata_size = _lagdata_size;
    memset(lagdata, 0, sizeof(float)*lagdata_size); // this could be removed once we're convinced thigs are working since the cuda memory is zeroed as well
    
    if (d_lagdata) hipFree(d_lagdata);
    hipMalloc((void**)&d_lagdata, lagdata_size * sizeof(float));
    hipMemset(d_lagdata, 0, lagdata_size * sizeof(float));
    
  }


}

void CUDA::CyclicFoldEngineCUDA::set_bin (uint64_t idat, double d_ibin, 
        double bins_per_sample)
{
	return;
	// idat ranges from idat_start to idat_start + binplansize
  unsigned ibin;
  int ilag;
  if ((last_ibin == nbin-1) && (last_ibin != int(d_ibin))) {
	  current_turn++;
  }
  for (ilag=0;ilag<nlag;ilag++) {
    ibin = unsigned(d_ibin + (((double)ilag)*bins_per_sample)/2.0); //half sample spacing
    unsigned ribin = ibin;
    unsigned planidx = current_turn*nbin*nlag + ibin*nlag + ilag;
    ibin = ibin % nbin; // ibin is wrapped phase
    if (lagbinplan[planidx].hits == 0) {
      lagbinplan[planidx].offset = idat;
      lagbinplan[planidx].ibin = ibin;
      lagbinplan[planidx].hits += 1;
    } 
    else {
      lagbinplan[planidx].hits += 1;
    }
//    cerr << "net:" << ribin + current_turn*nbin << " turn:" << current_turn << " ilag:" << ilag << " idx:" << planidx
//    		<< " hits:" << lagbinplan[planidx].hits << " offs:" << lagbinplan[planidx].offset << " idat:" << idat
//    		<< " ibin:" << ibin << " rbin:" << ribin << endl;
  }
  ndat_fold ++;
  last_ibin = int(d_ibin);
}

uint64_t CUDA::CyclicFoldEngineCUDA::get_bin_hits (int ibin)
{
	int iturn = 0;
	int idx = 0;
	idx = iturn*nbin*nlag + ibin*nlag; // we want the zero lag hits
	uint64_t hits = 0;
//	cerr << "ibin: " << ibin << " ";
	while (idx < binplan_size) {
		hits += lagbinplan[idx].hits;
		//cerr << lagbinplan[idx].hits << " ";
		iturn += 1;
		idx = iturn*nbin*nlag + ibin*nlag; // we want the zero lag hits
	}
//	cerr << "total: " << hits << endl;
	return hits;
}
  

__global__ void setBinsKernel(float phase_per_sample,
				float startph,
				float endph,
				unsigned intnturns,
				unsigned nlag,
                CUDA::bin* binplan)
{
	  unsigned ilaga = threadIdx.x;
	  unsigned nlaga = blockDim.x;
	  unsigned ilagb = blockIdx.x;
	  unsigned ibin = blockIdx.y;
	  unsigned nbin = gridDim.y;
	  unsigned ilag = ilagb*nlaga + ilaga;
	  float minph;
	  float maxph;
	  if (ilag >= nlag){
		  return;
	  }
	  unsigned planidx = nlag*ibin+ilag;
	  unsigned binplan_size = nlag*nbin*intnturns;
	  unsigned iturn = 0;
	  while(planidx < binplan_size) {
		  minph = (ibin*1.0)/nbin + iturn + (ilag*phase_per_sample)/2.0;
		  maxph = (ibin+1.0)/nbin + iturn + (ilag*phase_per_sample)/2.0;
		  maxph = min(endph,maxph); // same as if maxph > endph, maxph=endph
								// keep maxph from going off the end of the data block. In theory we should really pull more data from the next block, but for now
								// we just ignore correlations that span more than one data block

			if ((minph > endph) || (maxph < minph)) {
				// if the start of this lag/bin data is past the end of the data block (minph > endph), there is no valid data for this lag/bin
				// if maxph < minph, then it must be that minph > endph because the only way for this to happen would be if maxph were reassigned to endph in the previous clause.
				binplan[planidx].offset = 0;
				binplan[planidx].ibin = 0;
				binplan[planidx].hits = 0;
				continue;
			}

			if (minph > startph){
				// The basic case, the lag/bin data is fully within the data block, or goes right up to the end of the block (in which case maxph=endph)
				binplan[planidx].offset = round((minph-startph)/phase_per_sample);
				binplan[planidx].ibin = ibin;
				binplan[planidx].hits = round((maxph-minph)/phase_per_sample);
			}
			else if (maxph > startph){
				// In this case, the start of the lag/bin data precedes the first available data point, but there is still valid data from startph to maxph
			//					cerr << "minph < startph " << minph << " < " << startph << endl;
				binplan[planidx].offset = 0;
				binplan[planidx].ibin = ibin;
				binplan[planidx].hits = round((maxph-startph)/phase_per_sample);
			}
			else {
				// Finally, here minph <= startph and maxph <= startph, so the data needed fully precedes this data block.
			//					cerr << "maxph < startph " << minph << " < " << startph << endl;
				binplan[planidx].offset = 0;
				binplan[planidx].ibin = 0;
				binplan[planidx].hits = 0;
			}

			planidx += nlag*nbin;

	  }
}

// set_bins was added as a more efficient way of setting up the bin plan all in one go, rather than through repeated redundant calculations
// as was previously done using set_bin
// The bin plan is indexed as iturn*nbin*nlag + ibin*nlag + ilag
// each entry indicates the starting data sample (offset), the number of data samples to include in this lag/bin (hits), and the bin index (ibin)
// there is one entry for every lag, every bin, and for all turns in this data block.
uint64_t CUDA::CyclicFoldEngineCUDA::set_bins (double phi, double phase_per_sample, uint64_t _ndat, uint64_t idat_start)
{
	cerr << "Got to CUDA::CyclicFoldEngineCUDA::set_bins" << endl;


	phi = phi - floor(phi); // fractional phase at start
	double samples_per_bin = (1.0 / nbin) * (1.0 / phase_per_sample); // (1 turn / nbin bins) * (turns (phase) / sample) ^ -1
	double nturns = _ndat * phase_per_sample; // total number of turns represented by this block of data
	double minph,maxph;
	double startph = phi;  //starting fractional phase, the smallest valid phase
	double endph = startph + nturns; // final phase, the largest valid phase of any data point
	int startdat = 0;
	int intnturns = ceil(nturns) + 1;  // total number of turns in the binplan. This could probably be safely just ceil(nturns) but add 1 to be sure.
	int iturn,ibin,ilag;
	int planidx;

	int _binplan_size = intnturns*nbin*nlag; // total number of entries in the bin plan.

	ndat_fold = _ndat;

//	cerr << "Start ph:" << startph << " intnturns:" <<intnturns << " _ndat:" << _ndat << " nlag:" << nlag
//			<< " phase per sample:" << phase_per_sample<< " nturns:" << nturns << endl ;
//	cerr << "binplansize:" << binplan_size << "  _binplansize:" << _binplan_size << endl;

	// allocate memory for the binplan
	  if (_binplan_size > binplan_size) {

		    if (parent->verbose)
		      cerr << "dsp::CyclicFoldEngine::set_ndat alloc binplan" << endl;

		    if (lagbinplan) {
		      delete [] lagbinplan;
		    }
		    lagbinplan = new bin [_binplan_size];


		    binplan_size = _binplan_size;
		  }
	  memset(lagbinplan, 0 , sizeof(bin)*_binplan_size);  // all entries start out with zero hits, so any uninitialized portions will be ignored by the folding threads

#ifdef SET_BINS_GPU


	  uint64_t mem_size = binplan_size * sizeof(bin);
	  hipError_t error;

	  if (d_binplan == NULL) {
		  cerr << "setbinsgpu no binplan yet allocated" << endl;
	    error = hipMalloc ((void **)&(d_binplan),mem_size);
	    if (error != hipSuccess)
	        throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::set_bins_gpu",
	                     "hipMalloc orig %s %s",
	                     stream?"Async":"", hipGetErrorString (error));
	  } else {
		  // original plan was to check if binplan_size < orig_size so as to avoid extraneous free/malloc, but it
		  // seems that binplan_size gets reset each time before this funciton is called.
		  //cerr << "orig_size=" << orig_size << "< binplansize=" << binplan_size << "so freeing.." << endl;
		  error =hipFree(d_binplan);
		  if (error != hipSuccess)
			  throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::set_bins_gpu",
						   "hipFree %s %s",
						   stream?"Async":"", hipGetErrorString (error));
		  cerr << "realocating..." << endl;
		  error = hipMalloc ((void **)&(d_binplan),mem_size);
		  if (error != hipSuccess)
			  throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::set_bins_gpu",
						   "hipMalloc new %s %s",
						   stream?"Async":"", hipGetErrorString (error));
	  }
	  cerr << "now have d_binplan: " << d_binplan<< endl;
	  error = hipMemset(d_binplan, 0, mem_size);
	    if (error != hipSuccess)
	        throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::set_bins_gpu",
	                     "cudamemset orig %s %s",
	                     stream?"Async":"", hipGetErrorString (error));

	  const unsigned THREADS_PER_BLOCK = 1024;
	  unsigned nlaga,nlagb;
	  // if nlag*npol < THREADS_PER_BLOCK then nlaga = nlag, nlagb = 1
	  // else nlaga = THREADS_PER_BLOCK/npol, nlagb = nlag/nlaga + 1
	  if (nlag > THREADS_PER_BLOCK) {
		  nlaga = THREADS_PER_BLOCK;
		  nlagb = nlag/nlaga + 1;
	  }
	  else {
		  nlagb = 1;
		  nlaga = nlag;
	  }


	  dim3 blockDim (nlaga, 1, 1);
	  dim3 gridDim (nlagb, nbin, 1);
	  cerr << "setnlag=" << nlag;
	  cerr << "blockDim=" << blockDim.x << "," << blockDim.y << "," << blockDim.z << "," << endl;
	  cerr << "gridDim="  << gridDim.x << "," << gridDim.y << "," << gridDim.z << "," << endl;

	  unsigned lagbinplan_size = binplan_size;

	  setBinsKernel<<<gridDim,blockDim,0,stream>>>((float)phase_per_sample,
	  				(float)startph,
	  				(float)endph,
	  				intnturns,
	  				nlag,
	                d_binplan);

	  cerr << "finished setbins kernel" << endl;

	    cerr << "copying: stream=" << stream << " d_binplan=" << d_binplan << " mem_size=" << mem_size <<
	  		  " lagbinplan=" << lagbinplan << endl;
	    // Have to copy binplan back over to get hits on host
	    if (stream)
	      error = hipMemcpyAsync (lagbinplan,d_binplan,mem_size,hipMemcpyDeviceToHost,stream);
	    else
	      error = hipMemcpy (lagbinplan,d_binplan,mem_size,hipMemcpyDeviceToHost);
	    if (error != hipSuccess)
	      throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::set_binplan",
	                   "hipMemcpy%s %s",
	                   stream?"Async":"", hipGetErrorString (error));


#else


	for (iturn=0;iturn < intnturns; iturn++){
		for (ibin = 0; ibin < nbin; ibin++) {
			for (ilag=0; ilag < nlag; ilag++) {
				// minph is the starting phase of valid data for this lag/bin
				// maxph is the ending phase
				// thus we want to include all data points with phases in between minph and maxph
				minph = (ibin*1.0)/nbin + iturn + (ilag*phase_per_sample)/2.0;
				maxph = (ibin+1.0)/nbin + iturn + (ilag*phase_per_sample)/2.0;
				// index of this binplan entry
				planidx = iturn*nbin*nlag + ibin*nlag + ilag;

				if ( maxph > endph ) {
					maxph = endph; // keep maxph from going off the end of the data block. In theory we should really pull more data from the next block, but for now
									// we just ignore correlations that span more than one data block
				}
				if ((minph > endph) || (maxph < minph)) {
					// if the start of this lag/bin data is past the end of the data block (minph > endph), there is no valid data for this lag/bin
					// if maxph < minph, then it must be that minph > endph because the only way for this to happen would be if maxph were reassigned to endph in the previous clause.
					lagbinplan[planidx].offset = 0;
					lagbinplan[planidx].ibin = 0;
					lagbinplan[planidx].hits = 0;
					continue;
				}

				if (minph > startph){
					// The basic case, the lag/bin data is fully within the data block, or goes right up to the end of the block (in which case maxph=endph)
					lagbinplan[planidx].offset = round((minph-startph)/phase_per_sample);
					lagbinplan[planidx].ibin = ibin;
					lagbinplan[planidx].hits = round((maxph-minph)/phase_per_sample);
				}
				else if (maxph > startph){
					// In this case, the start of the lag/bin data precedes the first available data point, but there is still valid data from startph to maxph
//					cerr << "minph < startph " << minph << " < " << startph << endl;
					lagbinplan[planidx].offset = 0;
					lagbinplan[planidx].ibin = ibin;
					lagbinplan[planidx].hits = round((maxph-startph)/phase_per_sample);
				}
				else {
					// Finally, here minph <= startph and maxph <= startph, so the data needed fully precedes this data block.
//					cerr << "maxph < startph " << minph << " < " << startph << endl;
					lagbinplan[planidx].offset = 0;
					lagbinplan[planidx].ibin = 0;
					lagbinplan[planidx].hits = 0;
				}
				/*
				cerr << "iturn,ibin,ilag: " << iturn << "," << ibin << "," << ilag << ","
						<< " offset=" << lagbinplan[planidx].offset
						<< " hits=" << lagbinplan[planidx].hits
						<< " minph=" << minph
						<< " maxph=" << maxph

						<< endl;
				*/
			}
		}
	}
#endif
	cerr << "setbins returning iwth: " << ndat_fold;
	return ndat_fold;
}


void CUDA::CyclicFoldEngineCUDA::zero ()
{
  dsp::CyclicFoldEngine::zero();
  if (d_lagdata && lagdata_size>0) {
	  cerr << "CUDA::CyclicFoldEngineCUDA::zero: zeroing lagdata on gpu" << endl;
    hipMemset(d_lagdata, 0, lagdata_size * sizeof(float));
  }
}



void CUDA::CyclicFoldEngineCUDA::send_binplan ()
{

	/*
	 * current_turn is the highest number of turns that we needed in the set_bin stage
	 * so the total size of the binplan should be current_turn turns of nbin bins and nlag lags
	 * so we will update binplan_size accordingly
	 */
//	uint64_t orig_size = binplan_size;
//	binplan_size = (current_turn + 1) * nbin; // add one turn just for good measure. There should be zero hits in it
  uint64_t mem_size = binplan_size * sizeof(bin);

  if (dsp::Operation::verbose)
    cerr << "CUDA::CyclicFoldEngineCUDA::send_binplan ndat=" << ndat_fold 
         << "  Allocating on device mem_size " << mem_size
         << " binplan_size=" << binplan_size
         << " nlag=" << nlag
         << " sizeof(bin)=" << sizeof(bin)
         << " current_turn=" << current_turn
//         << " orig_size=" << orig_size
         << endl;

  hipError_t error;
#ifndef SET_BINS_GPU
  
  if (d_binplan == NULL) {
	  cerr << "no binplan yet allocated" << endl;
    error = hipMalloc ((void **)&(d_binplan),mem_size);
    if (error != hipSuccess)
        throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::send_binplan",
                     "hipMalloc orig %s %s",
                     stream?"Async":"", hipGetErrorString (error));
  } else {
	  // original plan was to check if binplan_size < orig_size so as to avoid extraneous free/malloc, but it
	  // seems that binplan_size gets reset each time before this funciton is called.
	  //cerr << "orig_size=" << orig_size << "< binplansize=" << binplan_size << "so freeing.." << endl;
	  error =hipFree(d_binplan);
	  if (error != hipSuccess)
		  throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::send_binplan",
					   "hipFree %s %s",
					   stream?"Async":"", hipGetErrorString (error));
	  cerr << "realocating..." << endl;
	  error = hipMalloc ((void **)&(d_binplan),mem_size);
	  if (error != hipSuccess)
		  throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::send_binplan",
					   "hipMalloc new %s %s",
					   stream?"Async":"", hipGetErrorString (error));
  }

/*  for (int k=binplan_size*nlag-nlag*16; k > 0; k -= nlag*nbin)
  {
	  if (lagbinplan[k].hits > 0){
		  cerr << "Found some hits at k = " << k << " = " << (k/(nlag*nbin)) << endl;
		  cerr << "current turn=" << current_turn << endl;
		  break;
	  }
  }*/

/*  ofstream fbin;
  fbin.open("cudabinplan.dat", ios::binary | ios::app);
  fbin.write((char *)(lagbinplan),mem_size);
  cerr << "done, dumping cudabinplan, closing files" << endl;
  fbin.close();
*/
//  cerr << "copying: stream=" << stream << " d_binplan=" << d_binplan << " mem_size=" << mem_size <<
//		  " lagbinplan=" << lagbinplan << endl;
  if (stream)
    error = hipMemcpyAsync (d_binplan,lagbinplan,mem_size,hipMemcpyHostToDevice,stream);
  else
    error = hipMemcpy (d_binplan,lagbinplan,mem_size,hipMemcpyHostToDevice);
  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::send_binplan",
                 "hipMemcpy%s %s", 
                 stream?"Async":"", hipGetErrorString (error));
#else
  cerr << "send_binplan: skipping since we directly set the bins on gpu" << endl;
#endif
}

void CUDA::CyclicFoldEngineCUDA::get_lagdata ()
{
  size_t lagdata_bytes = lagdata_size * sizeof(float);
  hipError_t error;
  if (stream) 
    error = hipMemcpyAsync (lagdata, d_lagdata, lagdata_bytes,
        hipMemcpyDeviceToHost, stream);
  else
    error = hipMemcpy (lagdata, d_lagdata, lagdata_bytes,
        hipMemcpyDeviceToHost);

  if (error != hipSuccess)
    throw Error (InvalidState, "CUDA::CyclicFoldEngineCUDA::get_lagdata",
                 "hipMemcpy%s %s", 
                 stream?"Async":"", hipGetErrorString (error));
}

/* 
 *  CUDA Kernels
 *
 */


// Since there is a maximum number of threads per block which may be less than the number of lags times number of pols,
// the ilag index is split into ilag = ilagb*nlaga + ilaga, where nlaga will be such that nlaga*npol = max_threads_per_block
// Each thread calculates the cyclic correlation for one lag for one bin for one input channel for one pol
// threadIdx.x -> ilaga    blockDim.x
// threadIdx.y -> pol
// threadIdx.z -> not used
// blockIdx.x -> ilagb
// blockIdx.y -> ibin
// blockIdx.z = ichan

// data is in FPT order, so chunks of time for a given pol and frequency
// in_span gives size of one time chunk for a given freq and pol in floats
__global__ void cycFoldIndPol (const float* in_base,
                unsigned in_span,
                float* out_base,
                unsigned binplan_size,
                unsigned nlag,
                CUDA::bin* binplan)
{
  unsigned ilaga = threadIdx.x;
  unsigned nlaga = blockDim.x;
  unsigned ilagb = blockIdx.x;
  unsigned ibin = blockIdx.y;
  unsigned ichan = blockIdx.z;
  unsigned ipol = threadIdx.y;
  unsigned npol = blockDim.y;
  unsigned nbin = gridDim.y;
  unsigned nchan = gridDim.z;
  unsigned ilag = ilagb*nlaga + ilaga;
  if (ilag >= nlag){
	  return;
  }
  unsigned planidx = nlag*ibin+ilag;
  const unsigned ndim = 2; // always complex data assumed

  if (planidx >= binplan_size) {
    return;
  }
  
  in_base  += in_span  * (ichan*npol + ipol);	//in_span is in units of float, so no need to mult by ndim
//  out_base += out_span * (ichan*npol + ipol);
  out_base += ndim*(ibin*npol*nchan*nlag
    + ipol*nchan*nlag
    + ichan*nlag 
    + ilag);
  
  unsigned bpstep = nlag*nbin; // step size to get to the next rotation for a given lag and bin in the binplan

  float2 total = make_float2(0.0,0.0);

  for (; planidx < binplan_size; planidx += bpstep)
  {
    const float* input = in_base + binplan[planidx].offset * ndim;
    const float* input2 = in_base + (binplan[planidx].offset + ilag) * ndim;
    const float2* a = (const float2*)(input);
    const float2* b = (const float2*)(input2);    

    for (unsigned i=0; i < binplan[planidx].hits; i++){
      total.x += a[i].x*b[i].x + a[i].y*b[i].y;
      total.y += a[i].y*b[i].x - a[i].x*b[i].y;
    }
  }

  out_base[0] += total.x;
  out_base[1] += total.y;
} 

void check_error (const char*);


void CUDA::CyclicFoldEngineCUDA::fold ()
{

  // TODO state/etc checks

  cerr << "In CyclicFoldEngineCUDA::fold d_binplan is: " << d_binplan << endl;
  setup ();
  send_binplan ();
  const unsigned THREADS_PER_BLOCK = 1024;
  unsigned nlaga,nlagb;
  // if nlag*npol < THREADS_PER_BLOCK then nlaga = nlag, nlagb = 1
  // else nlaga = THREADS_PER_BLOCK/npol, nlagb = nlag/nlaga + 1
  if (nlag*npol > THREADS_PER_BLOCK) {
	  nlaga = THREADS_PER_BLOCK/npol;
	  nlagb = nlag/nlaga + 1;
  }
  else {
	  nlagb = 1;
	  nlaga = nlag;
  }

  dim3 blockDim (nlaga, npol, 1);
  dim3 gridDim (nlagb, nbin, nchan);
  cerr << "nlag=" << nlag;
  cerr << "blockDim=" << blockDim.x << "," << blockDim.y << "," << blockDim.z << "," << endl;
  cerr << "gridDim="  << gridDim.x << "," << gridDim.y << "," << gridDim.z << "," << endl;
  
  unsigned lagbinplan_size = binplan_size;
  
  cycFoldIndPol <<<gridDim,blockDim,0,stream>>>(input,
                input_span,
                d_lagdata,
                lagbinplan_size,
                nlag,
                d_binplan);

  // profile on the device is no longer synchronized with the one on the host
  synchronized = false;

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error ("CUDA::CyclicFoldEngineCUDA::fold cuda error: ");
}

